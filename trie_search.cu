#include "hip/hip_runtime.h"
#include "trie_search.cuh"
#include "binary_trie.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "defines.h"

#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

__device__ void search(const unsigned int* input, const int* L, const int* R, const int numberOfSequences, const int sequenceLength, long long* matches, const int integer, const int bit, const int id)
{
	// look for a our sequence with one bit changed in the trie
	int ptr = 0;
	// unchanged bits on the left
	for (int i = 0; i < integer; i++)
	{
		for (int j = INTSIZE - 1; j >= 0; j--)
		{
			if ((input[i * numberOfSequences + id]) & (1 << j))
			{
				//if (id <= DEBUG_MAXID) printf("R");
				ptr = R[ptr];
			}

			else
			{
				//if (id <= DEBUG_MAXID) printf("L");
				ptr = L[ptr];
			}

			if (ptr == NOCHILD) return;
		}
	}
	//if (id <= debug_maxid) printf("same int, before change, id: %d\n", id);
	// same integer, before changed bit
	for (int j = INTSIZE - 1; j > bit; j--)
	{
		if ((input[integer * numberOfSequences + id]) & (1 << j))
		{
			//if (id <= DEBUG_MAXID) printf("R");
			ptr = R[ptr];
		}

		else
		{
			//if (id <= DEBUG_MAXID) printf("L");
			ptr = L[ptr];
		}

		if (ptr == NOCHILD) return;
	}
	//if (id <= DEBUG_MAXID) printf("bit change: %d, id: %d\n", bit, id);
	// bit change
	if ((input[integer * numberOfSequences + id]) & (1 << bit))
	{
		//if (id <= DEBUG_MAXID && bit == 0) printf("L, ptr: %d, id: %d\n",ptr, id);
		ptr = L[ptr];
	}
	else
	{
		//if (id <= DEBUG_MAXID && bit == 0) printf("R, ptr: %d, id :%d\n", ptr, id);
		ptr = R[ptr];
	}
	if (ptr == NOCHILD)
	{
		//if (id <= DEBUG_MAXID && bit == 0) printf("nochild:, id: %d\n", id);
		return;
	}

	// same integer, after changed bit
	for (int j = bit - 1; j >= 0; j--)
	{
		if ((input[integer * numberOfSequences + id]) & (1 << j))
		{
			//if (id <= DEBUG_MAXID) printf("Ri");
			ptr = R[ptr];
		}

		else
		{
			//if (id <= DEBUG_MAXID) printf("Li");
			ptr = L[ptr];
		}

		if (ptr == NOCHILD) return;
	}
	//unchanged bits on the right
	for (int i = integer + 1; i < sequenceLength; i++)
	{
		for (int j = INTSIZE - 1; j >= 0; j--)
		{
			if ((input[i * numberOfSequences + id]) & (1 << j))
			{
				//if (id <= DEBUG_MAXID) printf("Ru");
				ptr = R[ptr];
			}

			else
			{
				//if (id <= DEBUG_MAXID) printf("Lu");
				ptr = L[ptr];
			}

			if (ptr == NOCHILD) return;
		}
	}
	// found a match if we arrived at the end
	matches[id] += ptr;
}

__device__ void searchVerbose(const unsigned int* input, const int* L, const int* R, const int numberOfSequences, const int sequenceLength, long long* matches,
	const int integer, const int bit, const int id, const int printPairs, const int* minLRange, const int* minRRange)
{
	// look for a our sequence with one bit changed in the trie
	int ptr = 0;
	int prevPtr = 0;
	// unchanged bits on the left
	for (int i = 0; i < integer; i++)
	{
		for (int j = INTSIZE - 1; j >= 0; j--)
		{
			if ((input[i * numberOfSequences + id]) & (1 << j))
			{
				prevPtr = ptr;
				ptr = R[ptr];
			}

			else
			{
				prevPtr = ptr;
				ptr = L[ptr];
			}

			if (ptr == NOCHILD) return;
		}
	}
	//if (id <= debug_maxid) printf("same int, before change, id: %d\n", id);
	// same integer, before changed bit
	for (int j = INTSIZE - 1; j > bit; j--)
	{
		if ((input[integer * numberOfSequences + id]) & (1 << j))
		{
			prevPtr = ptr;
			ptr = R[ptr];
		}

		else
		{
			prevPtr = ptr;
			ptr = L[ptr];
		}

		if (ptr == NOCHILD) return;
	}
	//if (id <= DEBUG_MAXID) printf("bit change: %d, id: %d\n", bit, id);
	// bit change
	if ((input[integer * numberOfSequences + id]) & (1 << bit))
	{
		prevPtr = ptr;
		ptr = L[ptr];
	}
	else
	{
		prevPtr = ptr;
		ptr = R[ptr];
	}
	if (ptr == NOCHILD)
	{
		return;
	}

	// same integer, after changed bit
	for (int j = bit - 1; j >= 0; j--)
	{
		if ((input[integer * numberOfSequences + id]) & (1 << j))
		{
			prevPtr = ptr;
			ptr = R[ptr];
		}

		else
		{
			prevPtr = ptr;
			ptr = L[ptr];
		}

		if (ptr == NOCHILD) return;
	}
	//unchanged bits on the right
	for (int i = integer + 1; i < sequenceLength; i++)
	{
		for (int j = INTSIZE - 1; j >= 0; j--)
		{
			if ((input[i * numberOfSequences + id]) & (1 << j))
			{
				prevPtr = ptr;
				ptr = R[ptr];
			}

			else
			{
				prevPtr = ptr;
				ptr = L[ptr];
			}

			if (ptr == NOCHILD) return;
		}
	}
	// found a match if we arrived at the end
	matches[id] += ptr;
	//printf("matches[%d] = %d \n", id, matches[id] + ptr);
	// right
	int min;
	if (integer == sequenceLength - 1 && bit == 0)
		min = ((input[(sequenceLength - 1) * numberOfSequences + id]) & 1) ? minLRange[prevPtr] : minRRange[prevPtr];
	else
		min = ((input[(sequenceLength - 1) * numberOfSequences + id]) & 1) ? minRRange[prevPtr] : minLRange[prevPtr];
	for (int sequence = min; sequence <= min + ptr - 1; sequence++)
	{
		//if (sequence != id)
			printf("[%d, %d]\n", id, sequence);
	}
}

__global__ void searchKernel(const unsigned int* input, const int* L, const int* R, const int numberOfSequences, const int sequenceLength, long long* matches)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= numberOfSequences) return;
	
	matches[id] = 0;
	// for every bit
	for (int integer = 0; integer < sequenceLength; integer++)
	{
		for (int bit = INTSIZE - 1; bit >= 0; bit--)
		{
			search(input, L, R, numberOfSequences, sequenceLength, matches, integer, bit, id);
		}
	}
	//if (id <= DEBUG_MAXID) printf("matches[%d] = %lld\n", id, matches[id]);
}

__global__ void searchKernelVerbose(const unsigned int* input, const int* L, const int* R, const int numberOfSequences, const int sequenceLength, long long* matches,
	const int printPairs, const int* minLRange, const int* minRRange)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= numberOfSequences) return;
	matches[id] = 0;
	// for every bit
	for (int integer = 0; integer < sequenceLength; integer++)
	{
		for (int bit = INTSIZE - 1; bit >= 0; bit--)
		{
			searchVerbose(input, L, R, numberOfSequences, sequenceLength, matches, integer, bit, id, printPairs, minLRange, minRRange);
		}
	}
}

hipError_t allocateMemory(int** L, int** R, long long** matches, int** minLRange, int** minRRange, const int numberOfSequences, const int sequenceLength)
{
	hipError_t cudaStatus;
	long long memory = (long long)numberOfSequences * (long long)sequenceLength * (long long)32;

	cudaStatus = hipMalloc((void**)L, sizeof(int) * memory);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "L hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)R, sizeof(int) * memory);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "R hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)matches, sizeof(long long) * numberOfSequences);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)minLRange, sizeof(int) * memory);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "minLRange hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)minRRange, sizeof(int) * memory);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "minRRange hipMalloc failed!");
	}

	return cudaStatus;
}

hipError_t hammingOne(const unsigned int* input, const int numberOfSequences, const int sequenceLength, long long& matchCount, hipEvent_t& start, hipEvent_t& stop,
	float& totalTime, const int printPairs)
{
	hipError_t cudaStatus;
	int* L = 0;
	int* R = 0;
	int* minLRange = 0;
	int* minRRange = 0;
	long long* matches = 0;
	float milliseconds = 0;

	cudaStatus = allocateMemory(&L, &R, &matches, &minLRange, &minRRange, numberOfSequences, sequenceLength);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "allocateMemory failed!");
	}
	//build tree

	hipEventRecord(start);

	buildTrie(input, L, R, minLRange, minRRange, numberOfSequences, sequenceLength, printPairs);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaSynchronize failed!");
		goto Error;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totalTime += milliseconds;

	std::cout << "Building the tree completed. Time elapsed: " << milliseconds / 1000 << " seconds\n";

	// search for all possible sequences with Hamming distance equal to 1

	int threadsPerBlock = numberOfSequences < 1024 ? numberOfSequences : 1024;
	int blocks = (numberOfSequences + threadsPerBlock - 1) / threadsPerBlock;
	hipEventRecord(start);
	if (printPairs)
		searchKernelVerbose << <blocks, threadsPerBlock >> > (input, L, R, numberOfSequences, sequenceLength, matches,
			printPairs, minLRange, minRRange);
	else
		searchKernel << <blocks, threadsPerBlock >> > (input, L, R, numberOfSequences, sequenceLength, matches);

	thrust::device_ptr<long long> ptr = thrust::device_pointer_cast(matches);
	matchCount = thrust::reduce(ptr, ptr + numberOfSequences);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaSynchronize failed!");
		goto Error;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totalTime += milliseconds;

	std::cout << "Search completed. Time elapsed: " << milliseconds / 1000 << " seconds\n";

Error:
	hipFree(L);
	hipFree(R);
	hipFree(matches);
	hipFree(minLRange);
	hipFree(minRRange);
	return cudaStatus;
}