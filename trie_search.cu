#include "trie_search.cuh"
#include "binary_trie.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "defines.h"

#include <cstdlib>
#include <cstdio>
#include <iostream>

hipError_t hammingOne(const unsigned int* input, const int numberOfSequences, const int sequenceLength)
{
	hipError_t cudaStatus;
	int* L = 0;
	int* R = 0;

	long long memory = (long long)numberOfSequences * (long long)sequenceLength * (long long)32;

	cudaStatus = hipMalloc((void**)&L, sizeof(int) * memory);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&R, sizeof(int) * memory);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	printf("allocated %lld for L and R\n", memory * 32);

	//build tree
	buildTrie(input, L, R, numberOfSequences, sequenceLength);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaSynchronize failed!");
		goto Error;
	}

Error:
	hipFree(L);
	hipFree(R);
	return cudaStatus;
}