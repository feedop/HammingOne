#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#include "defines.h"
#include "CPU_binary_trie.hpp"

#include "radix_sort.cuh"
#include "trie_search.cuh"


#include <iostream>
#include <bitset>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <chrono>

void usage()
{
    fprintf(stderr, "USAGE: ./HammingOne [inputFile] [-c] [-v]\ne.g. ./HammingOne input.txt -c -v\n");
}

// reads integers from a file
int readFile(unsigned int** input, unsigned int** dev_input, FILE* file, int& sequenceLength, int& numberOfSequences)
{
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    char buf[BUFSIZE + 1];
    char* info;
    info = fgets(buf, BUFSIZE, file);
    int offset = strlen(info) + 1;
    if (info == NULL)
    {
        fprintf(stderr, "read error");
        return EXIT_FAILURE;
    }
    // get number of sequences
    info = strtok(info, ",");
    numberOfSequences = atoi(info);

    // get number of bits
    info = strtok(NULL, ",");
    int bits = atoi(info);
    sequenceLength = (bits + INTSIZE - 1) / INTSIZE;
    int remainder = bits % INTSIZE;

    // allocate memory for input
    *input = (unsigned int*)malloc(sizeof(int) * sequenceLength * numberOfSequences);

    hipError_t cudaStatus = hipMalloc((void**)dev_input, sizeof(int) * sequenceLength * numberOfSequences);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        return EXIT_FAILURE;
    }
    // read file to buffer
    fseek(file, offset, 0);
    char* bytes = (char*)malloc(numberOfSequences * (bits + 1) * sizeof(char));
    fread(bytes, sizeof(char), numberOfSequences * (bits + 1), file);
    // copy to input
    char temp[INTSIZE + 1];
    if (remainder == 0)
    {
        for (int i = 0; i < numberOfSequences; i++)
        {
            for (int j = 0; j < sequenceLength; j++)
            {
                memcpy(temp, bytes + i * (bits + 1) + j * INTSIZE, INTSIZE * sizeof(char));
                temp[INTSIZE] = '\0';
                (*input)[j * numberOfSequences + i] = strtoul(temp, NULL, 2);
            }
        }
    }
    else
    {
        for (int i = 0; i < numberOfSequences; i++)
        {
            // fill with leading zeros
            memcpy(temp, bytes + i * (bits + 1), remainder * sizeof(char));
            temp[remainder] = '\0';
            (*input)[i] = strtoul(temp, NULL, 2);
            for (int j = 1; j < sequenceLength; j++)
            {
                memcpy(temp, bytes + i * (bits + 1) + (j - 1) * INTSIZE + remainder, INTSIZE * sizeof(char));
                temp[INTSIZE] = '\0';
                (*input)[j * numberOfSequences + i] = strtoul(temp, NULL, 2);
            }
        }
    }
    free(bytes);
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Reading file complete. Time elapsed: " << std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / 1000000000.0 << " seconds" << std::endl;
    return EXIT_SUCCESS;
}

hipError_t generateRandoms(unsigned int* dev_input, const int sequenceLength, const int numberOfSequences);

// prints a few sequences from the beginning 
void head(unsigned int* input, const int sequenceLength, const int numberOfSequences)
{
    int vecLen = sequenceLength > 3 ? 3 : sequenceLength;
    int vectors = numberOfSequences > 10 ? 10 : numberOfSequences;
    std::vector<std::bitset<32>>bitset(vecLen * vectors);
    for (int i = 0; i < vecLen; i++)
    {
        for (int j = 0; j < vectors; j++)
        {
            bitset[i * vectors + j] = std::bitset<32>(input[i * numberOfSequences + j]);
        }
    }

    std::cout << "Values: \n";
    for (int i = 0; i < vectors; i++)
    {
        for (int j = 0; j < vecLen; j++)
        {
            std::cout << bitset[j * vectors + i] << " ";
        }
        std::cout << std::endl;
    }
}

__global__ void generateRandomsKernel(unsigned int *input, const int sequenceLength, const int numberOfSequences)
{
    unsigned int seed = blockIdx.x * blockDim.x + threadIdx.x;
    if (seed >= sequenceLength * numberOfSequences) return;
    hiprandState localState;
    hiprand_init(seed, 0, 0, &localState);
    input[seed] = hiprand(&localState);
}

int main(int argc, char* argv[])
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int sequenceLength = 0;
    int numberOfSequences = 0;
    float GPUmilliseconds = 0;
    float GPUtotalTime = 0;
    bool printPairs = false;
    bool compareToCPU = false;

    unsigned int* input = 0;
    unsigned int* dev_input = 0;;

    if (argc > 4 || argc == 1)
    {
        fprintf(stderr, "too many arguments ");
        usage();
        goto Error;
    }
    if (argc >= 2)
    {
        // read file
        FILE* file = fopen(argv[1], "r");
        if (!file)
        {
            fprintf(stderr, "no such file ");
            usage();
            fclose(file);
            goto Error;
        }

        if (readFile(&input, &dev_input, file, sequenceLength, numberOfSequences))
        {
            fclose(file);
            goto Error;
        }
    }
    if (argc >= 3)
    {   
        if (!strcmp(argv[2], "-c"))
            compareToCPU = true;
        else if (!strcmp(argv[2], "-v"))
            printPairs = true;
        else
        {
            usage();
            goto Error;
        }
    }
    if (argc == 4)
    {
        if (!strcmp(argv[3], "-c"))
            compareToCPU = true;
        else if (!strcmp(argv[3], "-v"))
            printPairs = true;
        else
        {
            usage();
            goto Error;
        }
    }

    // copy ints read from file to device
    cudaStatus = hipMemcpy(dev_input, input, sizeof(int) * sequenceLength * numberOfSequences, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // ---------------------------------------------------------------------------------

    // radix sort input
    hipEventRecord(start);
    cudaStatus = GPU::radixSort(&dev_input, sequenceLength, numberOfSequences);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "radixSort failed!");
        goto Error;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&GPUmilliseconds, start, stop);
    GPUtotalTime += GPUmilliseconds;

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(input, dev_input, sizeof(int) * sequenceLength * numberOfSequences, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    hipDeviceSynchronize();

    std::cout << "Radix sort completed. Time elapsed: " << GPUmilliseconds / 1000 << " seconds\n";

    // ---------------------------------------------------------------------------------

    // build a binary trie and search for all pairs with Hamming distance equal to 1
    long long matchCount = 0;

    cudaStatus = GPU::hammingOne(dev_input, numberOfSequences, sequenceLength, matchCount, start, stop, GPUtotalTime, printPairs);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    std::cout << " ======== GPU Results: ========\n" << "Finished. Total GPU time: " << GPUtotalTime / 1000 << " seconds\n";
    std::cout << "Matches found: " << matchCount << std::endl;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    // CPU algorithm
    if (compareToCPU)
    {
        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
        std::cout << " ======== CPU Results: ========\n";
        if (printPairs)
        {
            CPU::Trie<true> trie(input, sequenceLength, numberOfSequences);
            trie.hammingOne();
        }
        else
        {
            CPU::Trie<false> trie(input, sequenceLength, numberOfSequences);
            trie.hammingOne();
        }
        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        std::cout << "Finished. Total CPU time: " << std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / 1000000000.0 << " seconds" << std::endl;
    }
    
Error:
    free(input);
    hipFree(dev_input);
    //getchar();
    return 0;
}

// fills the input array with random unsigned ints
hipError_t generateRandoms(unsigned int* dev_input, const int sequenceLength, const int numberOfSequences)
{
    hipError_t cudaStatus;

    int threadsPerBlock = sequenceLength * numberOfSequences < 1024 ? sequenceLength * numberOfSequences : 1024;

    generateRandomsKernel << <(sequenceLength * numberOfSequences + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (dev_input, sequenceLength, numberOfSequences);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching generateRandomsKernel!\n", cudaStatus);
    }

    return cudaStatus;
}
