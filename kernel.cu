#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include "radix_sort.cuh"

#include <iostream>
#include <bitset>
#include <cstdio>
#include <cstdlib>
#include <vector>

hipError_t generateRandoms(unsigned int* dev_input, const int& sequenceLength, const int& numberOfSequences);

// prints a few sequences from the beginning 
void head(unsigned int* input, float milliseconds, const int& sequenceLength, const int& numberOfSequences)
{
    int vecLen = sequenceLength > 3 ? 3 : sequenceLength;
    int vectors = numberOfSequences > 10 ? 10 : numberOfSequences;
    std::vector<std::bitset<32>>bitset(vecLen * vectors);
    for (int i = 0; i < vecLen; i++)
    {
        for (int j = 0; j < vectors; j++)
        {
            bitset[i * vectors + j] = std::bitset<32>(input[(i + sequenceLength - vecLen) * numberOfSequences + j]);
        }
    }

    std::cout << "Values: \n";
    for (int i = 0; i < vectors; i++)
    {
        for (int j = vecLen - 1; j >= 0; j--)
        {
            std::cout << bitset[j * vectors + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << "Time elapsed: " << milliseconds / 1000 << " seconds\n";
}

__global__ void generateRandomsKernel(unsigned int *input, const int sequenceLength, const int numberOfSequences)
{
    unsigned int seed = blockIdx.x * blockDim.x + threadIdx.x;
    if (seed >= sequenceLength * numberOfSequences) return;
    hiprandState localState;
    hiprand_init(seed, 0, 0, &localState);
    input[seed] = hiprand(&localState);
}

int main()
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int sequenceLength = 32;
    int numberOfSequences = 100000;    

    unsigned int* input = (unsigned int*)malloc(sizeof(int) * sequenceLength * numberOfSequences);
    unsigned int* dev_input = 0;

    cudaStatus = hipMalloc((void**)&dev_input, sizeof(int) * sequenceLength * numberOfSequences);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // generate random sequences
    hipEventRecord(start);
    cudaStatus = generateRandoms(dev_input, sequenceLength, numberOfSequences);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(input, dev_input, sizeof(int) * sequenceLength * numberOfSequences, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // print
    head(input, milliseconds, sequenceLength, numberOfSequences);

    // ---------------------------------------------------------------------------------

    // radix sort input
    hipEventRecord(start);
    cudaStatus = radixSort(&dev_input, sequenceLength, numberOfSequences);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(input, dev_input, sizeof(int) * sequenceLength * numberOfSequences, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // print
    head(input, milliseconds, sequenceLength, numberOfSequences);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
Error:
    free(input);
    hipFree(dev_input);
    //getchar();
    return 0;
}

// fills the input array with random unsigned ints
hipError_t generateRandoms(unsigned int* dev_input, const int& sequenceLength, const int& numberOfSequences)
{
    hipError_t cudaStatus;

    int threadsPerBlock = sequenceLength * numberOfSequences < 1024 ? sequenceLength * numberOfSequences : 1024;

    generateRandomsKernel << <(sequenceLength * numberOfSequences + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (dev_input, sequenceLength, numberOfSequences);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching generateRandomsKernel!\n", cudaStatus);
    }

    return cudaStatus;
}
